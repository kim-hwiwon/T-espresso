#include "hip/hip_runtime.h"
#include "../lib/common.h"

extern "C" {

  
/**************************************************** 
 *  void ___cuprof_trace();
 *
 *  Write trace data and associated info
 *  to the externally allocated areas.
 */
  __device__ __noinline__ void ___cuprof_trace(uint32_t* alloc, uint32_t* commit,
                                               uint32_t* flushed, uint32_t* signal,
                                               uint8_t* records, uint64_t addr,
                                               uint64_t grid, uint64_t ctaid_serial,
                                               uint32_t warpv, uint32_t lane,
                                               uint32_t instid, uint32_t kernid,
                                               uint32_t sm, uint32_t warpp,
                                               uint16_t req_size,
                                               uint8_t type, uint8_t to_be_traced) {
    
    if (!to_be_traced)
      return;

    uint64_t clock;
    asm volatile ("mov.u64 %0, %%clock64;" : "=l"(clock));
    uint32_t active;
    asm volatile ("activemask.b32 %0;" : "=r"(active));
    uint32_t lowest = __ffs(active)-1;
    uint32_t rlane_id = __popc(active << (32 - lane));

    volatile uint32_t* alloc_v = alloc;
    volatile uint32_t* commit_v = commit;
    volatile uint32_t* flushed_v = flushed;
    volatile uint32_t* signal_v = signal;
    
    uint32_t rec_offset;
    
    
    // allocate space in slot
    if (lane == lowest) {


      //uint32_t counter = 0;
      do {
        while (*alloc_v >= RECORDS_PER_SLOT) {
          //counter++;
          //if ((counter & 0xFFFFF) == 0xFFFFF)
          //  printf("%u (%u)\n", *alloc_v, counter++);
        }

        //printf("%u, %u, %u\n", *alloc_v, *commit_v, *signal_v);//////////////
      } while ((rec_offset = atomicInc(alloc, UINT32_MAX)) >= RECORDS_PER_SLOT);

      // write header at lowest lane
      /*
      record_header_t* rec_header =
        (record_header_t*) &(records[rec_offset * RECORD_SIZE]);
      *rec_header =
        (record_header_t) RECORD_SET_INIT_OPT(0, type, instid, kernid, warpv,
                                              ctaid_serial,
                                              grid,
                                              warpp, sm,
                                              req_size, clock);
      */
      //printf("WRITTEN (%u)\n", alloc_raw);//////////////////////////////
    }


    // write requested addr for each lane
    
    rec_offset = __shfl_sync(active, rec_offset, lowest);
    //if (lane == lowest) {
    
    uint64_t* rec_base = (uint64_t*) &(records[(rec_offset) * RECORD_SIZE +
                                               (rlane_id*24)]);
    rec_base[0] = ctaid_serial;
    rec_base[1] = grid;
    rec_base[2] = addr;
    
    

    // guarantee all writes before to be written to the 'records'
    __threadfence_system();
    
    // commit space in slot, and send full signal to the host
    if (lane == lowest) {
      uint32_t commit_raw = atomicInc(commit, UINT32_MAX) + 1;
      //printf("end (%u / %u)\n", commit_raw, *flushed_v); ////////////////////////
      if (commit_raw == RECORDS_PER_SLOT) {
        //printf("signaled! (%u)\n", RECORDS_PER_SLOT);/////////////////////////////
        //*signal_v = commit_raw; // request flush to host
        *signal_v = RECORDS_PER_SLOT;
        //printf("DEV_commit_v: %u\n", *commit_v);
        //__threadfence_system();
        //*flushed_v = UINT32_MAX; // request sent successfully
      }
    }
    
  }

  

/**************************************************** 
 *  void ___cuprof_trace_ret();
 *
 *  Flush commit_v to signal (host)
 */
  __device__ void ___cuprof_trace_ret(uint32_t* commit, uint32_t* signal,
                                      uint32_t lane) {
    
    uint32_t active;
    asm volatile ("activemask.b32 %0;" : "=r"(active));
    uint32_t lowest = __ffs(active)-1;

    volatile uint32_t* commit_v = commit;

    if (lane == lowest) {
      __threadfence();
      //printf("ret\n");//////////////////
      uint32_t rec_count = *commit_v;

      // if request not sent at the point of return, then send request
      if (rec_count != UINT32_MAX) {
        atomicMax(signal, rec_count); /////////////// need to be fixed
      }
      // guarantee write before return
      __threadfence_system();
    }

  }

  

/**************************************************** 
 *  void ___cuprof_filter();
 *
 *  Check if current thread is to be traced, 
 *  with given thread-constant vars (grid, cta, warpv).
 *
 *  Called only once in a thread, when the thread starts.
 */
  __device__ void ___cuprof_filter(uint8_t* to_be_traced, uint64_t* filter_grid,
                                   uint64_t* filter_cta, uint32_t* filter_warpv,
                                   uint8_t filter_grid_count,
                                   uint8_t filter_cta_count,
                                   uint8_t filter_warpv_count,
                                   uint64_t ctaid_serial, uint32_t warpv) {
    
    uint64_t grid;
    asm volatile ("mov.u64 %0, %%gridid;" : "=l"(grid));

    // count == 0 (do not filter): default value is true (!0)
    // otherwise: default value is false (!count), and set to true if exists in filter
    uint8_t to_be_traced_per_type[3] = {
      !filter_grid_count,
      !filter_cta_count,
      !filter_warpv_count
    };
    

    // check grid filter
    for (uint32_t i = 0; i < filter_grid_count; i++)
      if (filter_grid[i] == grid)
        to_be_traced_per_type[0] = 1;

    // check cta filter
    for (uint32_t i = 0; i < filter_cta_count; i++)
      if (filter_cta[i] == ctaid_serial)
        to_be_traced_per_type[1] = 1;

    // check warpv filter
    for (uint32_t i = 0; i < filter_warpv_count; i++)
      if (filter_warpv[i] == warpv)
        to_be_traced_per_type[2] = 1;

    // combine per_type with AND conditions
    uint8_t result = 1;
    for (uint32_t i = 0; i < 3; i++)
      if (!to_be_traced_per_type[i])
        result = 0;

    *to_be_traced = result;
  }


  
/**************************************************** 
 *  void ___cuprof_filter_volatile();
 *
 *  Check if current thread is to be traced, 
 *  with given volatile vars (sm, warpp).
 *
 *  Called on every trace, iff the filter of sm, warpp is set.
 */
  __device__ void ___cuprof_filter_volatile(uint8_t* to_be_traced,
                                            uint32_t* filter_sm, uint32_t* filter_warpp,
                                            uint8_t filter_sm_count,
                                            uint8_t filter_warpp_count,
                                            uint32_t sm, uint32_t warpp) {

    // count == 0 (do not filter): default value is true (!0)
    // otherwise: default value is false (!count), and set to true if exists in filter
    uint8_t to_be_traced_per_type[2] = {
      !filter_sm_count,
      !filter_warpp_count
    };
    
    // check sm filter
    for (uint32_t i = 0; i < filter_sm_count; i++)
      if (filter_sm[i] == sm)
        to_be_traced_per_type[0] = 1;

    // check warpp filter
    for (uint32_t i = 0; i < filter_warpp_count; i++)
      if (filter_warpp[i] == warpp)
        to_be_traced_per_type[1] = 1;

    // combine per_type with AND conditions
    uint8_t result = 1;
    for (uint32_t i = 0; i < 2; i++)
      if (!to_be_traced_per_type[i])
        result = 0;

    *to_be_traced = result;
  }
}
