#include "hip/hip_runtime.h"
#include "../lib/common.h"

extern "C" {


  
  
#define DEBUG_PRINT {                                                   \
    char str_filter_raw[36] = {};                                       \
    char str_filter_raw_prev_lanes[36] = {};                            \
    char str_filter[36] = {};                                           \
    char str_filter_prev_lanes[36] = {};                                \
                                                                        \
    uint32_t* uint_ptr;                                                 \
    char* str_ptr;                                                      \
                                                                        \
    uint32_t* uint_ptr_list[] = {                                       \
      &filter_raw,                                                      \
      &filter_raw_prev_lanes,                                           \
      &filter,                                                          \
      &filter_prev_lanes                                                \
    };                                                                  \
    char* str_ptr_list[] = {                                            \
      str_filter_raw,                                                   \
      str_filter_raw_prev_lanes,                                        \
      str_filter,                                                       \
      str_filter_prev_lanes                                             \
    };                                                                  \
                                                                        \
    for (int i = 0;                                                     \
         i < sizeof(uint_ptr_list) / sizeof(uint_ptr_list[0]);          \
         i++) {                                                         \
      uint_ptr = uint_ptr_list[i];                                      \
      str_ptr = str_ptr_list[i];                                        \
      for (int i = 0; i < 35; i++) {                                    \
        if (i == 8 || i == 17 || i == 26)                               \
          str_ptr[i] = ' ';                                             \
        else                                                            \
          str_ptr[i] = (*uint_ptr & (0x1 << (i - (i / 9)) ) ?'1':'0');  \
      }                                                                 \
    }                                                                   \
                                                                        \
                                                                        \
    printf("\n"                                                         \
           "lane:             \t%u\n"                                   \
           "addr_prev_prev:   \t%08lX\n"                                \
           "addr_prev:        \t%08lX\n"                                \
           "addr_delta_prev:  \t%08lX\n"                                \
           "addr_delta:       \t%08lX\n"                                \
           "is_delta_changed: \t%c\n"                                   \
           "is_prev_inactive: \t%c\n"                                   \
           "is_write_rawf:    \t%c\n"                                   \
           "filter_raw:       \t%s\n"                                   \
           "filter_raw_prevs: \t%s\n"                                   \
           "consec_write:     \t%u\n"                                   \
           "is_write:         \t%c\n"                                   \
           "filter:           \t%s\n"                                   \
           "filter_prevs:     \t%s\n"                                   \
           "write_pos:        \t%u\n"                                   \
           "write_count:      \t%u\n"                                   \
           "record_size:      \t%lu\n\n\n"                              \
           ,                                                            \
           lane,                                                        \
           addr_prev_prev,                                              \
           addr_prev,                                                   \
           addr_delta_prev,                                             \
           addr_delta,                                                  \
           is_delta_changed?'O':'X',                                    \
           is_prev_inactive?'O':'X',                                    \
           is_write_rawf?'O':'X',                                       \
           str_filter_raw,                                              \
           str_filter_raw_prev_lanes,                                   \
           consec_write,                                                \
           is_write?'O':'X',                                            \
           str_filter,                                                  \
           str_filter_prev_lanes,                                       \
           write_pos,                                                   \
           write_count,                                                 \
           record_size                                                  \
      );                                                                \
  }

  
  

/****************************************************
 *  void ___cuprof_trace();
 *
 *  Write trace data and associated info
 *  to the externally allocated areas.
 */
  __device__ __noinline__ void ___cuprof_trace(uint32_t* alloc, uint32_t* commit,
                                               uint32_t* flushed, uint32_t* signal,
                                               uint8_t* records, uint64_t addr,
                                               uint64_t grid, uint64_t ctaid_serial,
                                               uint32_t warpv, uint32_t lane,
                                               uint32_t instid, uint32_t kernid,
                                               uint32_t sm, uint32_t warpp,
                                               uint8_t to_be_traced) {

    if (!to_be_traced)
      return;

    uint64_t clock;
    asm volatile ("mov.u64 %0, %%clock64;" : "=l"(clock));

    volatile uint32_t* alloc_v = alloc;
    volatile uint32_t* commit_v = commit;
    volatile uint32_t* flushed_v = flushed;
    volatile uint32_t* signal_v = signal;
    
    uint32_t active = __activemask();
    uint32_t lowest = __ffs(active)-1;

    uint32_t lanemask = (0x1 << lane);
    uint32_t lanemask_prevs = lanemask - 1;
    uint32_t rlane_id = __popc(active & lanemask_prevs);
    uint32_t n_active = __popc(active);

    uint32_t rec_offset;
    uint32_t flushed_cur;




    
    uint32_t msb = (uint32_t)(addr >> 32);
    int is_msb_same = 1;
    //uint64_t msb = addr & 0xFFFFFFFF00000000;
    //int addr_len;
    //__match_all_sync(active, msb, &addr_len);

    // addr delta mask
    

    uint64_t addr_prev_prev = __shfl_up_sync(active, addr, 2);
    uint64_t addr_prev = __shfl_up_sync(active, addr, 1);
    //uint64_t addr_next = __shfl_down_sync(active, addr, 1);
    //uint64_t addr_delta_prev_prev = addr_prev - addr_prev_prev;
    uint64_t addr_delta_prev = addr_prev - addr_prev_prev;
    uint64_t addr_delta = addr - addr_prev;
    //int is_delta_changed_prev = (addr_delta_prev_prev != addr_delta_prev);
    uint32_t is_delta_changed = (addr_delta != addr_delta_prev);
    //int is_write_f1 = is_delta_changed && is_delta_changed_prev;
    uint32_t is_prev_inactive = (~(active << 1) & lanemask);
    uint32_t is_write_rawf = is_delta_changed | is_prev_inactive;
  
    uint32_t filter_raw = __ballot_sync(active, is_write_rawf);
  
    //uint32_t is_inactive_prev = active << 1;
    //uint32_t is_inactive_next = active >> 1;
    //uint32_t is_prev_set = subfilter_1 << 1;
    //uint32_t inactive_no_write = 0xFFFFFFFF; //is_inactive_next | is_prev_set;
    //uint32_t inactive_force_write = ~is_inactive_prev;
  
    //uint32_t subfilter_2 = (subfilter_1 | inactive_force_write) & active;
  
    uint32_t filter_raw_prev_lanes = filter_raw << (32-1 - lane);
    uint32_t consec_write = __clz(~filter_raw_prev_lanes);
    uint32_t is_write = consec_write & 0x1;
  
    uint32_t filter = __ballot_sync(active, is_write);
  
  
    uint32_t filter_prev_lanes = filter & lanemask_prevs;
    uint8_t write_pos = __popc(filter_prev_lanes);
    uint8_t write_count = __popc(filter);
    uint64_t record_size = RECORD_SIZE(write_count);


    

    //DEBUG_PRINT;


    //////////////////////////
    /*
    uint32_t is_write = 1; //consec_write & 0x1;
    
    uint32_t filter = 0xFFFFFFFF;

    
    uint8_t write_pos = lane; //__popc(prev_lanes_write_mask);
    uint8_t write_count = 32; //__popc(filter);
    uint64_t record_size = RECORD_SIZE(write_count);
    */


    // allocate space in slot
    if (lane == lowest) {

      // get the allocated offset
      uint32_t alloc_raw = atomicAdd(alloc, record_size); //atomicInc(alloc, UINT32_MAX);
      rec_offset = alloc_raw % SLOT_SIZE;
      //rec_offset = alloc_raw & (RECORDS_PER_SLOT-1);

      // wait until slot is not full
      do {
        flushed_cur = *flushed_v;
      } while ((alloc_raw - flushed_cur) >= SLOT_SIZE - RECORD_MAX_SIZE);


      //volatile uint64_t* rec_header = (uint64_t*) &(records[rec_offset]);
      //for (int i = 0; i < write_count + RECORD_HEADER_UNIT; i++)
      //  while (rec_header[0]);
      //printf("%u\n", rec_offset);

      // map alloc to physical buf
      //rec_offset = alloc_raw - flushed_now;

      ////////// WRITE DISTRIBUTION (OFF) //////////

      // write header at lowest lane
      /*
      record_header_t* rec_header =
        (record_header_t*) &(records[rec_offset * RECORD_MAX_SIZE]);

      *rec_header =
        (record_header_t) RECORD_SET_INIT_OPT(0, type, instid, kernid, warpv,
                                              ctaid_serial,
                                              grid,
                                              warpp, sm,
                                              req_size, clock);
      */
      //////////////////////////////////////////////
    }

    //uint64_t msb = addr & 0xFFFFFFFF00000000;
    //int addr_len;
    //__match_all_sync(active, msb, &addr_len);
    

    ////////// WRITE DISTRIBUTION (ON) //////////

    // write header

    //__match_all_sync(active, msb, &pred);

    rec_offset = __shfl_sync(active, rec_offset, lowest);    

    uint64_t header_info[RECORD_HEADER_UNIT];
    header_info[0] = RECORD_SET_INIT_IDX_0(0, instid, kernid, warpv) | 1;
    header_info[1] = RECORD_SET_INIT_IDX_1(ctaid_serial) | 1;
    header_info[2] = RECORD_SET_INIT_IDX_2(grid) | 1;
    header_info[3] = RECORD_SET_INIT_IDX_3(warpp, sm) | 1;
    header_info[4] = RECORD_SET_INIT_IDX_4(clock) | 1;
    header_info[5] = (is_msb_same ? RECORD_SET_INIT_IDX_5(addr, active) : 0) | 1;


    volatile uint64_t* rec_header = (uint64_t*) (records + rec_offset);

    for (int i = rlane_id; i < RECORD_HEADER_UNIT; i += n_active) {
      int rec_i = (rec_offset + sizeof(uint64_t)*i) % SLOT_SIZE;
      *(uint64_t*)(records + rec_i) = header_info[i];
    }

    //////////////////////////////////////////////


    






    /////////////////////////////////////////////


    //uint32_t filter_final = active;

    //uint32_t write_pos = lane;
    //uint32_t is_write = 1;
    
    // write reqeusted addrs for each lane
    if (is_write) {
      int rec_i = (rec_offset + RECORD_SIZE(write_pos)) % SLOT_SIZE;
      volatile uint64_t* rec_addr = (uint64_t*) (records + rec_i);
      *rec_addr = (uint64_t) addr | 1;
    }



    // guarantee all writes before to be written to the 'records'
    //__threadfence_system();

    // commit space in slot, and send full signal to the host
    if (lane == lowest) {
      //uint32_t write_count = __popc(filter_final);
      //if (write_count != 32)
      //printf("write_count = %u\n%x\n%x\n%u\n%u\n%u\n\n%x\n%x\n%x)\n", write_count, addr_prev, addr_next, addr_delta_prev, addr_delta_next, is_delta_changed, subfilter_1, subfilter_2, filter);//////////////////
      uint32_t commit_raw = atomicAdd(commit, record_size) + record_size; //atomicInc(commit, UINT32_MAX) + 1;
      
      
      //if (commit_raw - flushed_cur >= SLOT_SIZE - RECORD_MAX_SIZE) { //(commit_raw & ((RECORDS_PER_SLOT-1))) == 0) {
      uint32_t flush_unit = UNIT_SLOT_SIZE - RECORD_MAX_SIZE;
      uint32_t flush_threshold = UNIT_SLOT_SIZE - (2*RECORD_MAX_SIZE);
      if (
        ( ((commit_raw - flushed_cur - record_size) % flush_unit) >= flush_threshold )
        &&
        ( (commit_raw - flushed_cur) % flush_unit < flush_threshold )
        ) {
        *signal_v = commit_raw; // request flush to host
        //printf("%u - %u (%u)\n", flushed_cur, commit_raw, commit_raw - flushed_cur);///////
      //__threadfence_system();
      }
    }

  }



/****************************************************
 *  void ___cuprof_trace_ret();
 *
 *  Flush commit_v to signal (host)
 */
  __device__ void ___cuprof_trace_ret(uint32_t* commit, uint32_t* signal,
                                      uint32_t lane) {
/*
    return;

    uint32_t active;
    asm volatile ("activemask.b32 %0;" : "=r"(active));
    uint32_t lowest = __ffs(active)-1;

    volatile uint32_t* commit_v = commit;

    if (lane == lowest) {
      //__threadfence();
      //printf("ret\n");//////////////////
      //uint32_t rec_count = *commit_v;

      // if request not sent at the point of return, then send request
      //if (rec_count != UINT32_MAX) {
      
      //atomicMax(signal, *commit_v); /////////////// need to be fixed
      // Bug possibility: what if signal overflows?
      
        //}
      // guarantee write before return
      //__threadfence_system();
    }
*/
  }



/****************************************************
 *  void ___cuprof_filter();
 *
 *  Check if current thread is to be traced,
 *  with given thread-constant vars (grid, cta, warpv).
 *
 *  Called only once in a thread, when the thread starts.
 */
  __device__ void ___cuprof_filter(uint8_t* to_be_traced, uint64_t* filter_grid,
                                   uint64_t* filter_cta, uint32_t* filter_warpv,
                                   uint8_t filter_grid_count,
                                   uint8_t filter_cta_count,
                                   uint8_t filter_warpv_count,
                                   uint64_t ctaid_serial, uint32_t warpv) {

    uint64_t grid;
    asm volatile ("mov.u64 %0, %%gridid;" : "=l"(grid));

    // count == 0 (do not filter): default value is true (!0)
    // otherwise: default value is false (!count), and set to true if exists in filter
    uint8_t to_be_traced_per_type[3] = {
      !filter_grid_count,
      !filter_cta_count,
      !filter_warpv_count
    };


    // check grid filter
    for (uint32_t i = 0; i < filter_grid_count; i++)
      if (filter_grid[i] == grid)
        to_be_traced_per_type[0] = 1;

    // check cta filter
    for (uint32_t i = 0; i < filter_cta_count; i++)
      if (filter_cta[i] == ctaid_serial)
        to_be_traced_per_type[1] = 1;

    // check warpv filter
    for (uint32_t i = 0; i < filter_warpv_count; i++)
      if (filter_warpv[i] == warpv)
        to_be_traced_per_type[2] = 1;

    // combine per_type with AND conditions
    uint8_t result = 1;
    for (uint32_t i = 0; i < 3; i++)
      if (!to_be_traced_per_type[i])
        result = 0;

    *to_be_traced = result;
  }



/****************************************************
 *  void ___cuprof_filter_volatile();
 *
 *  Check if current thread is to be traced,
 *  with given volatile vars (sm, warpp).
 *
 *  Called on every trace, iff the filter of sm, warpp is set.
 */
  __device__ void ___cuprof_filter_volatile(uint8_t* to_be_traced,
                                            uint32_t* filter_sm, uint32_t* filter_warpp,
                                            uint8_t filter_sm_count,
                                            uint8_t filter_warpp_count,
                                            uint32_t sm, uint32_t warpp) {

    // count == 0 (do not filter): default value is true (!0)
    // otherwise: default value is false (!count), and set to true if exists in filter
    uint8_t to_be_traced_per_type[2] = {
      !filter_sm_count,
      !filter_warpp_count
    };

    // check sm filter
    for (uint32_t i = 0; i < filter_sm_count; i++)
      if (filter_sm[i] == sm)
        to_be_traced_per_type[0] = 1;

    // check warpp filter
    for (uint32_t i = 0; i < filter_warpp_count; i++)
      if (filter_warpp[i] == warpp)
        to_be_traced_per_type[1] = 1;

    // combine per_type with AND conditions
    uint8_t result = 1;
    for (uint32_t i = 0; i < 2; i++)
      if (!to_be_traced_per_type[i])
        result = 0;

    *to_be_traced = result;
  }


  
}
