#include "hip/hip_runtime.h"
#include "../lib/Common.h"
#include "../lib/TraceIO.h"

#include <atomic>
#include <string>
#include <thread>
#include <vector>

#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <libgen.h>


#define always_assert(cond) do {                                        \
    if (!(cond)) {                                                      \
      printf("assertion failed at %s:%d: %s\n", __FILE__, __LINE__, #cond); \
      abort();                                                          \
    }                                                                   \
  } while(0)

#define cudaChecked(code) do {                                  \
    hipError_t err = code;                                     \
    if (err != hipSuccess) {                                   \
      printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__,   \
             hipGetErrorString(err));                          \
      abort();                                                  \
    }                                                           \
  } while(0)

extern "C" {
  static char* ___cuprof_accdat_var = NULL;
  static uint64_t ___cuprof_accdat_varlen = 0;
}

static const char* getexename() {
  static char* cmdline = NULL;

  if (cmdline != NULL) {
    return cmdline;
  }

  FILE* file = fopen("/proc/self/cmdline", "r");
  if (!file) {
    return NULL;
  }
  size_t n;
  getdelim(&cmdline, &n, 0, file);
  fclose(file);
  cmdline = basename(cmdline);
  return cmdline;
}

/** Allows to specify base name for traces. The first occurence of
 * "?" is replaced with an ID unique to each stream.
 * Default pattern: "./trace-?.bin"
 */
static std::string traceName(std::string id) {
  static const char* exename = getexename(); // initialize once
  const char* pattern_env = getenv("CUPROF_TRACE_PATTERN");
  std::string pattern;
  if (pattern_env) {
    pattern = pattern_env;
  } else if (exename) {
    pattern = "./trace-" + std::string(exename) + "-?.trc";
  } else {
    pattern = "./trace-?.trc";
  }

  size_t pos = pattern.find("?");
  if (pos != std::string::npos) {
    pattern.replace(pos, 1, id);
  }
  return pattern;
}

/*******************************************************************************
 * TraceConsumer sets up and consumes a queue that can be used by kernels to
 * to write their traces into.
 * Only correct when accessed by a single cuda stream.
 * Usage must follow a strict protocol:
 * - one call to TraceConsumer()
 * - zero or more calls to start() ALWAYS followed by stop()
 * - one call to ~TraceConsumer()
 * Trying to repeatedly start or stop a consumer results in process termination.
 *
 * The queue is a multiple producer, single consumer key. Circular queues do not
 * work as expected because we cannot reliably update the in-pointer with a single
 * atomic operation. The result would be corrupted data as the host begins reading
 * data that is falsely assumed to have been committed.
 *
 * Instead we use buffers that are alternatingly filled up by the GPU and cleared
 * out by the CPU.
 * Two pointers are associated with each buffer, an allocation and a commit pointer.
 * A GPU warp first allocates spaces in the buffer using an atomic add on the
 * allocation pointer, then writes its data and increases the commit buffer by the
 * same amount, again using atomic add.
 * The buffered is considered full 
 * a) by the GPU if the allocation pointer is within 32 elements of capacity, and
 * b) by the host if the commit pointer is within 32 elements of capacity.
 * When the buffer is full, all elements are read by the host and the commit and
 * allocation buffer are reset to 0 in this order.
 * 
 * Since a maximum of 1 warp is writing some of the last 32 elements, the commit
 * pointer pointing in this area signals that all warps have written their data.
 * 
 * Several buffers, called "slots", exist in order to reduce contention.
 *
 * Allocation and commit pointers are uint32_t with 64 Byte padding to avoid cache thrashing.
 */


typedef struct kernel_trace_arg_t {
  const char* kernel_name;
  uint16_t kernel_block_size;
} kernel_trace_arg_t;

typedef struct trace_filter_arg_t {
  const uint8_t* sm;
  const uint64_t* cta;
  const uint32_t* warp;
  size_t sm_size, cta_size, warp_size;
} trace_filter_arg_t;


class TraceConsumer {
public:
  static char* debugdata;
  
  //TraceConsumer(std::string suffix, const char* header_info) {
  TraceConsumer(std::string suffix, trace_filter_arg_t trace_filter) {

    //printf("___CUDATRACE_DEBUG_DATA"); //
    //printf(" (%p) = ", ___CUDATRACE_DEBUG_DATA); //
    //printf("%s\n", ___CUDATRACE_DEBUG_DATA); //
    
    this->suffix = suffix;
    TraceConsumer::trace_filter = trace_filter;

    cudaChecked(hipHostAlloc(&records_host, SLOTS_NUM * SLOTS_SIZE * RECORD_SIZE, hipHostMallocMapped));
    cudaChecked(hipHostGetDevicePointer(&records_device, records_host, 0));

    cudaChecked(hipHostAlloc(&allocs_host, SLOTS_NUM * CACHELINE, hipHostMallocMapped));
    cudaChecked(hipHostGetDevicePointer(&allocs_device, allocs_host, 0));
    memset(allocs_host, 0, SLOTS_NUM * CACHELINE);

    cudaChecked(hipHostAlloc(&commits_host, SLOTS_NUM * CACHELINE, hipHostMallocMapped));
    cudaChecked(hipHostGetDevicePointer(&commits_device, commits_host, 0));
    memset(commits_host, 0, SLOTS_NUM * CACHELINE);

    should_run = false;
    does_run = false;

    pipe_name = traceName(suffix);

    output = fopen(this->pipe_name.c_str(), "wb");
    if (output == nullptr) {
      printf("unable to open trace file '%s' for writing\n", pipe_name.c_str());
      abort();
    }

    trace_write_header(output, ___cuprof_accdat_var, ___cuprof_accdat_varlen);
  }

  virtual ~TraceConsumer() {
    always_assert(!should_run);
    fclose(output);

    hipHostFree(records_host);
    hipHostFree(allocs_host);
    hipHostFree(commits_host);
  }

  void start(const char* name, uint16_t block_size) {
    always_assert(!should_run);
    should_run = true;

    // reset all buffers and pointers
    memset(allocs_host, 0, SLOTS_NUM * CACHELINE);
    memset(commits_host, 0, SLOTS_NUM * CACHELINE);
    // just for testing purposes
    memset(records_host, 0, SLOTS_NUM * SLOTS_SIZE * RECORD_SIZE);

    trace_write_kernel(output, name, block_size);

    worker_thread = std::thread(consume, this);

    while (!does_run) {}
  }

  void stop() {
    always_assert(should_run);
    should_run = false;
    while (does_run) {}
    worker_thread.join();
  }

  void fillTraceinfo(traceinfo_t* info) {
    info->allocs = allocs_device;
    info->commits = commits_device;
    info->records = records_device;
    info->slot_size = SLOTS_SIZE;
  }

protected:

  static uint64_t rdtsc(){
    unsigned int lo, hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
  }

  // clear up a slot if it is full
  static int consumeSlot(uint8_t* alloc_ptr, uint8_t* commit_ptr, uint8_t* records_ptr,
                         FILE* out, bool is_kernel_active, trace_record_t* acc) {
    // allocs/commits is written by threads on the GPU, so we need it volatile
    volatile uint32_t* vcommit = (uint32_t*)commit_ptr;
    volatile uint32_t* valloc = (uint32_t*)alloc_ptr;

    
    // flush only if kernel is not active,
    // or kernel is active but slot is full and all allocs are committed (no writes anymore)
    uint32_t records_count = *vcommit;
    if (is_kernel_active &&
        (records_count <= SLOTS_SIZE - 32 || records_count != *valloc)) {
      return 1;
    }


    // compression mode
    // mode = 1 : same addresses
    // mode = 2 : increment addresses
    //            (before_addr + before_size * before_count = cur_addr)
    int compression_mode;
    char newrec_orig[TRACE_RECORD_SIZE(32)] = {0};
    trace_record_t* const newrec = (trace_record_t* const) newrec_orig;

    
    trace_record_addr_t* acc_addr;
    // we know writing from the gpu stopped, so we avoid using the volatile
    // reference in the end condition
    for (int32_t i = 0; i < records_count; ++i) {

      trace_deserialize((record_t*)&records_ptr[i * RECORD_SIZE], newrec);


      // trace filter
      
      bool to_be_traced = true;
      
      if (trace_filter.sm_size > 0) {
        
        bool is_found = false;
        for (int i = 0; i < trace_filter.sm_size; i++)
          if (trace_filter.sm[i] == newrec->sm)
            is_found = true;
        
        if (!is_found)
          to_be_traced = false;
      }
      
      if (trace_filter.cta_size > 0) {
        
        bool is_found = false;
        for (int i = 0; i < trace_filter.cta_size; i++)
          if ( trace_filter.cta[i] ==
               ((((uint64_t)newrec->ctaid.x) << 32) |
                (((uint64_t)newrec->ctaid.y & 0xFFFF) << 16) |
                ((uint64_t)newrec->ctaid.z & 0xFFFF)) )
            is_found = true;
        
        if (!is_found)
          to_be_traced = false;
      }
      
      if (trace_filter.warp_size > 0) {
        
        bool is_found = false;
        for (int i = 0; i < trace_filter.warp_size; i++)
          if (trace_filter.warp[i] == newrec->warp_v)
            is_found = true;
        
        if (!is_found)
          to_be_traced = false;
      }


      if (!to_be_traced)
        continue;
      

      
      // if this is the first record, intialize it
      if (acc->addr_unit->count == 0) {
	memcpy(acc, newrec, sizeof(trace_record_t));
        //*acc = *newrec;
	acc->addr_len = 1;
        acc_addr = &acc->addr_unit[acc->addr_len - 1];
	acc->addr_unit->count = 1;
	acc->addr_unit->addr = newrec->addr_unit->addr;
	compression_mode = 0;
      }

      
      // otherwise see if we can increment or have to flush
      else {

        // set compression info on second record of the addr_unit
        if (acc_addr->count == 1) {
          int64_t offset = (int64_t)newrec->addr_unit->addr - (int64_t)acc_addr->addr;
          if ((offset & 0xFFFFFFFFFF000000) == 0 ||
              (offset & 0xFFFFFFFFFF000000) == 0xFFFFFFFFFF000000) {
            acc_addr->offset = (int32_t) (offset & 0xFFFFFFFF);
            compression_mode = 1;
          }
        }


        // if same inst info with the record before - to be compressed
        if (newrec->type == acc->type && newrec->req_size == acc->req_size &&
	    newrec->grid == acc->grid && newrec->ctaid.x == acc->ctaid.x &&
	    newrec->ctaid.y == acc->ctaid.y && newrec->ctaid.z == acc->ctaid.z &&
	    newrec->warp_v == acc->warp_v && newrec->clock == acc->clock) {

          // same inst info & addr pattern - increment current addr_unit count
          if ( (compression_mode == 1 && newrec->addr_unit->addr == acc_addr->addr +
                (acc_addr->offset * acc_addr->count)) ) {
            acc_addr->count += 1;
          }

          // same inst info but new addr pattern - add new addr_unit
          else {
            acc->addr_len += 1;
            acc_addr = &acc->addr_unit[acc->addr_len - 1];
            acc_addr->count = 1;
            acc_addr->addr = newrec->addr_unit->addr;
            compression_mode = 0;
          }

        }


        // if different inst info - add new inst info
        else {
          trace_write_record(out, acc);
          
          memcpy(acc, newrec, TRACE_RECORD_SIZE(1));
          acc->addr_len = 1;
          acc_addr = &acc->addr_unit[acc->addr_len - 1];
          acc->addr_unit->count = 1;
          acc->addr_unit->addr = newrec->addr_unit->addr;
          compression_mode = 0;
        }
          
      }
    }

    *vcommit = 0;
    // ensure commits are reset first
    std::atomic_thread_fence(std::memory_order_release);
    *valloc = 0;

    return 0;
  }

  // payload function of queue consumer
  static void consume(TraceConsumer* obj) {
    obj->does_run = true;
    
    char record_acc_orig[TRACE_RECORD_SIZE(32)] = {0};
    trace_record_t* const record_acc = (trace_record_t* const) record_acc_orig;
    // record_acc->addr_len == 0 -> uninitialized

    uint8_t* allocs = obj->allocs_host;
    uint8_t* commits = obj->commits_host;
    uint8_t* records = obj->records_host;

    FILE* sink = obj->output;


    while(obj->should_run) {
      for(int slot = 0; slot < SLOTS_NUM; slot++) {
        uint32_t allocs_offset = slot * CACHELINE;
        uint32_t commits_offset = slot * CACHELINE;
        uint32_t records_offset = slot * SLOTS_SIZE * RECORD_SIZE;
        consumeSlot(&allocs[allocs_offset], &commits[commits_offset],
                    &records[records_offset], sink, true, record_acc);
      }
    }

    // after should_run flag has been reset to false, no warps are writing, but
    // there might still be data in the buffers
    for(int slot = 0; slot < SLOTS_NUM; slot++) {
      uint32_t allocs_offset = slot * CACHELINE;
      uint32_t commits_offset = slot * CACHELINE;
      uint32_t records_offset = slot * SLOTS_SIZE * RECORD_SIZE;
      consumeSlot(&allocs[allocs_offset], &commits[commits_offset],
                  &records[records_offset], sink, false, record_acc);
    }

    // flush accumulator and reset to uninitialized (if at all initialized)
    if (record_acc->addr_len > 0) {
      trace_write_record(sink, record_acc);
      record_acc->addr_len = 0;
    }

    obj->does_run = false;
    return;
  }

  std::string suffix;
  static trace_filter_arg_t trace_filter;

  std::atomic<bool> should_run;
  std::atomic<bool> does_run;

  FILE* output;
  std::thread       worker_thread;
  std::string       pipe_name;

  uint8_t* allocs_host, * allocs_device;
  uint8_t* commits_host, * commits_device;
  uint8_t* records_host, * records_device;
};
trace_filter_arg_t TraceConsumer::trace_filter;

/*******************************************************************************
 * TraceManager acts as a cache for TraceConsumers and ensures only one consumer
 * per stream is exists. RAII on global variable closes files etc.
 * CUDA API calls not allowed inside of stream callback, so TraceConsumer
 * initialization must be performed explicitly;
 */
class TraceManager {
public:
  /** Creates a new consumer for a stream if necessary. Returns true if a new
   * consumer had to be created, false otherwise.
   */
  //bool touchConsumer(hipStream_t stream, const char* header_info) {
  bool touchConsumer(hipStream_t stream, trace_filter_arg_t trace_filter) {
    for (auto &consumer_pair : consumers) {
      if (consumer_pair.first == stream) {
        return false;
      }
    }

    char* suffix;
    asprintf(&suffix, "%d", (int)consumers.size());
    auto new_pair = std::make_pair(stream, new TraceConsumer(suffix, trace_filter));
    free(suffix);
    consumers.push_back(new_pair);
    return true;
  }

  /** Return *already initialized* TraceConsumer for a stream. Aborts application
   * if stream is not initialized.
   */
  TraceConsumer* getConsumer(hipStream_t stream) {
    for (auto &consumer_pair : consumers) {
      if (consumer_pair.first == stream) {
        return consumer_pair.second;
      }
    }
    always_assert(0 && "trying to get non-existent consumer");
    return nullptr;
  }

  virtual ~TraceManager() {
    for (auto &consumer_pair : consumers) {
      delete consumer_pair.second;
    }
  }
private:
  std::vector<std::pair<hipStream_t, TraceConsumer*>> consumers;
};
char* TraceConsumer::debugdata = nullptr;

TraceManager __trace_manager;

/*******************************************************************************
 * C Interface
 */

extern "C" {
  
  void ___cuprof_accdat_ctor() {
    if (!___cuprof_accdat_var) {
      ___cuprof_accdat_var = (char*) malloc(sizeof(char));
    }
  }
  
  void ___cuprof_accdat_dtor() {
    if (___cuprof_accdat_var) {
      free(___cuprof_accdat_var);
      ___cuprof_accdat_var = NULL;
    }
  }

  void ___cuprof_accdat_append(const char* data, uint64_t data_len) {
    char* var_tmp = (char*) realloc(___cuprof_accdat_var,
                                    ___cuprof_accdat_varlen + data_len + 1);
    if (!var_tmp) {
      fprintf(stderr, "cuprof: Failed to initialize memory access data!\n");
      abort();
    }

    memcpy(var_tmp + ___cuprof_accdat_varlen, data, data_len);
    var_tmp[___cuprof_accdat_varlen + data_len] = '\0';
    
    ___cuprof_accdat_var = var_tmp;
    ___cuprof_accdat_varlen += data_len;
  }


  
  static void __trace_start_callback(hipStream_t stream, hipError_t status, void* vargs) {
    auto* consumer = __trace_manager.getConsumer(stream);
    kernel_trace_arg_t* vargs_cast = (kernel_trace_arg_t*)vargs;
    consumer->start(vargs_cast->kernel_name, vargs_cast->kernel_block_size);
    free(vargs_cast);
  }

  static void __trace_stop_callback(hipStream_t stream, hipError_t status, void* vargs) {
    auto* consumer = __trace_manager.getConsumer(stream);
    consumer->stop();
  }

  
  
  void __trace_fill_info(const void* info, hipStream_t stream) {
    auto* consumer = __trace_manager.getConsumer(stream);
    consumer->fillTraceinfo((traceinfo_t*) info);
  }

  void __trace_copy_to_symbol(hipStream_t stream, const void* symbol, const void* info) {
    cudaChecked(hipMemcpyToSymbolAsync(HIP_SYMBOL(symbol), info, sizeof(traceinfo_t), 0, hipMemcpyHostToDevice, stream));
  }

  void __trace_touch(hipStream_t stream,
                     uint8_t* sm_filter, uint64_t* cta_filter, uint32_t* warp_filter,
                     size_t sm_filter_size, size_t cta_filter_size, size_t warp_filter_size) {
    
    trace_filter_arg_t filter = {
      sm_filter, cta_filter, warp_filter,
      sm_filter_size, cta_filter_size, warp_filter_size
    };
    
    __trace_manager.touchConsumer(stream, filter);
  }

  void __trace_start(hipStream_t stream, const char* kernel_name, uint16_t block_size) {
    kernel_trace_arg_t* arg = (kernel_trace_arg_t*) malloc(sizeof(kernel_trace_arg_t));
    if (arg == nullptr) {
      printf("unable to allocate memory\n");
      abort();
    }
    
    *arg = (kernel_trace_arg_t){kernel_name, block_size};
    cudaChecked(hipStreamAddCallback(stream,
                                      __trace_start_callback, (void*)arg, 0));
  }

  void __trace_stop(hipStream_t stream) {
    cudaChecked(hipStreamAddCallback(stream,
                                      __trace_stop_callback, (void*)nullptr, 0));
  }

}
