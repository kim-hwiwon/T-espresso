#include "hip/hip_runtime.h"
#include "../lib/Common.h"
#include "../lib/cutrace_io.h"

#include <atomic>
#include <string>
#include <thread>
#include <vector>

#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <libgen.h>


#define always_assert(cond) do {                                        \
    if (!(cond)) {                                                      \
      printf("assertion failed at %s:%d: %s\n", __FILE__, __LINE__, #cond); \
      abort();                                                          \
    }                                                                   \
  } while(0)

#define cudaChecked(code) do {                                  \
    hipError_t err = code;                                     \
    if (err != hipSuccess) {                                   \
      printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__,   \
             hipGetErrorString(err));                          \
      abort();                                                  \
    }                                                           \
  } while(0)

extern "C" {
  static char * ___cuprof_accdat_var = NULL;
  static uint64_t ___cuprof_accdat_varlen = 0;
}

static const char* getexename() {
  static char* cmdline = NULL;

  if (cmdline != NULL) {
    return cmdline;
  }

  FILE *f = fopen("/proc/self/cmdline", "r");
  if (!f) {
    return NULL;
  }
  size_t n;
  getdelim(&cmdline, &n, 0, f);
  fclose(f);
  cmdline = basename(cmdline);
  return cmdline;
}

/** Allows to specify base name for traces. The first occurence of
 * "?" is replaced with an ID unique to each stream.
 * Default pattern: "./trace-?.bin"
 */
static std::string traceName(std::string id) {
  static const char* exename = getexename(); // initialize once
  const char* pattern_env = getenv("MEMTRACE_PATTERN");
  std::string pattern;
  if (pattern_env) {
    pattern = pattern_env;
  } else if (exename) {
    pattern = "./" + std::string(exename) + "-?.trc";
  } else {
    pattern = "./trace-?.trc";
  }

  size_t pos = pattern.find("?");
  if (pos != std::string::npos) {
    pattern.replace(pos, 1, id);
  }
  return pattern;
}

/*******************************************************************************
 * TraceConsumer sets up and consumes a queue that can be used by kernels to
 * to write their traces into.
 * Only correct when accessed by a single cuda stream.
 * Usage must follow a strict protocol:
 * - one call to TraceConsumer()
 * - zero or more calls to start() ALWAYS followed by stop()
 * - one call to ~TraceConsumer()
 * Trying to repeatedly start or stop a consumer results in process termination.
 *
 * The queue is a multiple producer, single consumer key. Circular queues do not
 * work as expected because we cannot reliably update the in-pointer with a single
 * atomic operation. The result would be corrupted data as the host begins reading
 * data that is falsely assumed to have been committed.
 *
 * Instead we use buffers that are alternatingly filled up by the GPU and cleared
 * out by the CPU.
 * Two pointers are associated with each buffer, an allocation and a commit pointer.
 * A GPU warp first allocates spaces in the buffer using an atomic add on the
 * allocation pointer, then writes its data and increases the commit buffer by the
 * same amount, again using atomic add.
 * The buffered is considered full 
 * a) by the GPU if the allocation pointer is within 32 elements of capacity, and
 * b) by the host if the commit pointer is within 32 elements of capacity.
 * When the buffer is full, all elements are read by the host and the commit and
 * allocation buffer are reset to 0 in this order.
 * 
 * Since a maximum of 1 warp is writing some of the last 32 elements, the commit
 * pointer pointing in this area signals that all warps have written their data.
 * 
 * Several buffers, called "slots", exist in order to reduce contention.
 *
 * Allocation and commit pointers are uint32_t with 64 Byte padding to avoid cache thrashing.
 */


typedef struct kernel_trace_arg_t {
  const char * kernel_name;
  uint16_t kernel_block_size;
} kernel_trace_arg_t;

typedef struct trace_filter_arg_t {
  const uint8_t* sm;
  const uint64_t* cta;
  const uint32_t* warp;
  const size_t* size;
} trace_filter_arg_t;

//extern const char ___CUDATRACE_DEBUG_DATA[];

class TraceConsumer {
public:
  static char * debugdata;
  
  //TraceConsumer(std::string suffix, const char* header_info) {
  TraceConsumer(std::string suffix, trace_filter_arg_t trace_filter) {

    //printf("___CUDATRACE_DEBUG_DATA"); //
    //printf(" (%p) = ", ___CUDATRACE_DEBUG_DATA); //
    //printf("%s\n", ___CUDATRACE_DEBUG_DATA); //
    
    this->suffix = suffix;
    TraceConsumer::trace_filter = trace_filter;

    cudaChecked(hipHostAlloc(&RecordsHost, SLOTS_NUM * SLOTS_SIZE * RECORD_SIZE, hipHostMallocMapped));
    cudaChecked(hipHostGetDevicePointer(&RecordsDevice, RecordsHost, 0));

    cudaChecked(hipHostAlloc(&AllocsHost, SLOTS_NUM * CACHELINE, hipHostMallocMapped));
    cudaChecked(hipHostGetDevicePointer(&AllocsDevice, AllocsHost, 0));
    memset(AllocsHost, 0, SLOTS_NUM * CACHELINE);

    cudaChecked(hipHostAlloc(&CommitsHost, SLOTS_NUM * CACHELINE, hipHostMallocMapped));
    cudaChecked(hipHostGetDevicePointer(&CommitsDevice, CommitsHost, 0));
    memset(CommitsHost, 0, SLOTS_NUM * CACHELINE);

    shouldRun = false;
    doesRun = false;

    pipeName = traceName(suffix);

    output = fopen(this->pipeName.c_str(), "wb");
    if (output == nullptr) {
      printf("unable to open trace file '%s' for writing\n", pipeName.c_str());
      abort();
    }

    trace_write_header(output, ___cuprof_accdat_var, ___cuprof_accdat_varlen);
  }

  virtual ~TraceConsumer() {
    always_assert(!shouldRun);
    fclose(output);

    hipHostFree(RecordsHost);
    hipHostFree(AllocsHost);
    hipHostFree(CommitsHost);
  }

  void start(const char *name, uint16_t block_size) {
    always_assert(!shouldRun);
    shouldRun = true;

    // reset all buffers and pointers
    memset(AllocsHost, 0, SLOTS_NUM * CACHELINE);
    memset(CommitsHost, 0, SLOTS_NUM * CACHELINE);
    // just for testing purposes
    memset(RecordsHost, 0, SLOTS_NUM * SLOTS_SIZE * RECORD_SIZE);

    trace_write_kernel(output, name, block_size);

    workerThread = std::thread(consume, this);

    while (!doesRun) {}
  }

  void stop() {
    always_assert(shouldRun);
    shouldRun = false;
    while (doesRun) {}
    workerThread.join();
  }

  void fillTraceinfo(traceinfo_t *info) {
    info->allocs = AllocsDevice;
    info->commits = CommitsDevice;
    info->records = RecordsDevice;
    info->slot_size = SLOTS_SIZE;
  }

protected:

  static uint64_t rdtsc(){
    unsigned int lo,hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
  }

  // clear up a slot if it is full
  static int consumeSlot(uint8_t *allocPtr, uint8_t *commitPtr, uint8_t *recordsPtr,
                         FILE* out, bool kernelActive, trace_record_t *acc) {
    // allocs/commits is written by threads on the GPU, so we need it volatile
    volatile uint32_t *vcommit = (uint32_t*)commitPtr;
    volatile uint32_t *valloc = (uint32_t*)allocPtr;

    
    // if kernel is still active we only want to read full slots
    uint32_t numRecords = *vcommit;
    if (kernelActive && !(numRecords > SLOTS_SIZE - 32)) {
      return 1;
    }


    // compression mode
    // mode = 1 : same addresses
    // mode = 2 : increment addresses
    //            (before_addr + before_size * before_count = cur_addr)
    int compression_mode;
    char newrecOrig[TRACE_RECORD_SIZE(32)] = {0};
    trace_record_t *const newrec = (trace_record_t *const) newrecOrig;

    
    trace_record_addr_t *acc_addr;
    // we know writing from the gpu stopped, so we avoid using the volatile
    // reference in the end condition
    for (int32_t i = 0; i < numRecords; ++i) {

      __trace_unpack((record_t *)&recordsPtr[i * RECORD_SIZE], newrec);


      // filter
      
      bool to_be_traced = true;
      
      if (trace_filter.size[0] > 0) {
        
        bool is_found = false;
        for (int i = 0; i < trace_filter.size[0]; i++)
          if (trace_filter.sm[i] == newrec->smid)
            is_found = true;
        
        if (!is_found)
          to_be_traced = false;
      }
      
      if (trace_filter.size[1] > 0) {
        
        bool is_found = false;
        for (int i = 0; i < trace_filter.size[1]; i++)
          if ( trace_filter.cta[i] ==
              ((((uint64_t)newrec->ctaid.x) << 32) |
              (((uint64_t)newrec->ctaid.y & 0xFFFF) << 16) |
               ((uint64_t)newrec->ctaid.z & 0xFFFF)) )
            is_found = true;
        
        if (!is_found)
          to_be_traced = false;
      }
      
      if (trace_filter.size[2] > 0) {
        
        bool is_found = false;
        for (int i = 0; i < trace_filter.size[2]; i++)
          if (trace_filter.warp[i] == newrec->warp)
            is_found = true;
        
        if (!is_found)
          to_be_traced = false;
      }


      if (!to_be_traced)
        continue;
      

      
      // if this is the first record, intialize it
      if (acc->addr_unit->count == 0) {
	memcpy(acc, newrec, sizeof(trace_record_t));
        //*acc = *newrec;
	acc->addr_len = 1;
        acc_addr = &acc->addr_unit[acc->addr_len - 1];
	acc->addr_unit->count = 1;
	acc->addr_unit->addr = newrec->addr_unit->addr;
	compression_mode = 0;
      }

      // otherwise see if we can increment or have to flush
      else {

        // set compression info on second record of the addr_unit
        if (acc_addr->count == 1) {
          int64_t offset = (int64_t)newrec->addr_unit->addr - (int64_t)acc_addr->addr;
          if ((offset & 0xFFFFFFFFFF000000) == 0 ||
              (offset & 0xFFFFFFFFFF000000) == 0xFFFFFFFFFF000000) {
            acc_addr->offset = (int32_t) (offset & 0xFFFFFFFF);
            compression_mode = 1;
          }
        }


        // if same inst info with the record before - to be compressed
        if (newrec->type == acc->type && newrec->size == acc->size &&
	    newrec->smid == acc->smid && newrec->ctaid.x == acc->ctaid.x &&
	    newrec->ctaid.y == acc->ctaid.y && newrec->ctaid.z == acc->ctaid.z &&
	    newrec->warp == acc->warp && newrec->clock == acc->clock) {

          // same inst info & addr pattern - increment current addr_unit count
          if ( (compression_mode == 1 && newrec->addr_unit->addr == acc_addr->addr +
                (acc_addr->offset * acc_addr->count)) ) {
            acc_addr->count += 1;
          }

          // same inst info but new addr pattern - add new addr_unit
          else {
            acc->addr_len += 1;
            acc_addr = &acc->addr_unit[acc->addr_len - 1];
            acc_addr->count = 1;
            acc_addr->addr = newrec->addr_unit->addr;
            compression_mode = 0;
          }

        }


        // if different inst info - add new inst info
        else {
          trace_write_record(out, acc);
          
          memcpy(acc, newrec, TRACE_RECORD_SIZE(1));
          acc->addr_len = 1;
          acc_addr = &acc->addr_unit[acc->addr_len - 1];
          acc->addr_unit->count = 1;
          acc->addr_unit->addr = newrec->addr_unit->addr;
          compression_mode = 0;
        }
          
      }
    }

    *vcommit = 0;
    // ensure commits are reset first
    std::atomic_thread_fence(std::memory_order_release);
    *valloc = 0;

    return 0;
  }

  // payload function of queue consumer
  static void consume(TraceConsumer *obj) {
    obj->doesRun = true;
    
    char recordAccOrig[TRACE_RECORD_SIZE(32)] = {0};
    trace_record_t *const recordAcc = (trace_record_t *const) recordAccOrig;
    // recordAcc->addr_len == 0 -> uninitialized

    uint8_t *allocs = obj->AllocsHost;
    uint8_t *commits = obj->CommitsHost;
    uint8_t *records = obj->RecordsHost;

    FILE* sink = obj->output;


    while(obj->shouldRun) {
      for(int slot = 0; slot < SLOTS_NUM; slot++) {
        uint32_t allocs_offset = slot * CACHELINE;
        uint32_t commits_offset = slot * CACHELINE;
        uint32_t records_offset = slot * SLOTS_SIZE * RECORD_SIZE;
        consumeSlot(&allocs[allocs_offset], &commits[commits_offset],
                    &records[records_offset], sink, true, recordAcc);
      }
    }

    // after shouldRun flag has been reset to false, no warps are writing, but
    // there might still be data in the buffers
    for(int slot = 0; slot < SLOTS_NUM; slot++) {
      uint32_t allocs_offset = slot * CACHELINE;
      uint32_t commits_offset = slot * CACHELINE;
      uint32_t records_offset = slot * SLOTS_SIZE * RECORD_SIZE;
      consumeSlot(&allocs[allocs_offset], &commits[commits_offset],
                  &records[records_offset], sink, false, recordAcc);
    }

    // flush accumulator and reset to uninitialized (if at all initialized)
    if (recordAcc->addr_len > 0) {
      trace_write_record(sink, recordAcc);
      recordAcc->addr_len = 0;
    }

    obj->doesRun = false;
    return;
  }

  std::string suffix;
  static trace_filter_arg_t trace_filter;

  std::atomic<bool> shouldRun;
  std::atomic<bool> doesRun;
  //trace_record_t recordAcc; // recordAccumulator for compression

  FILE *output;
  std::thread       workerThread;
  std::string       pipeName;

  uint8_t *AllocsHost, *AllocsDevice;
  uint8_t *CommitsHost, *CommitsDevice;
  uint8_t *RecordsHost, *RecordsDevice;
};
trace_filter_arg_t TraceConsumer::trace_filter;

/*******************************************************************************
 * TraceManager acts as a cache for TraceConsumers and ensures only one consumer
 * per stream is exists. RAII on global variable closes files etc.
 * CUDA API calls not allowed inside of stream callback, so TraceConsumer
 * initialization must be performed explicitly;
 */
class TraceManager {
public:
  /** Creates a new consumer for a stream if necessary. Returns true if a new
   * consumer had to be created, false otherwise.
   */
  //bool touchConsumer(hipStream_t stream, const char* header_info) {
  bool touchConsumer(hipStream_t stream, trace_filter_arg_t trace_filter) {
    for (auto &consumerPair : consumers) {
      if (consumerPair.first == stream) {
        return false;
      }
    }

    char *suffix;
    asprintf(&suffix, "%d", (int)consumers.size());
    auto newPair = std::make_pair(stream, new TraceConsumer(suffix, trace_filter));
    free(suffix);
    consumers.push_back(newPair);
    return true;
  }

  /** Return *already initialized* TraceConsumer for a stream. Aborts application
   * if stream is not initialized.
   */
  TraceConsumer *getConsumer(hipStream_t stream) {
    for (auto &consumerPair : consumers) {
      if (consumerPair.first == stream) {
        return consumerPair.second;
      }
    }
    always_assert(0 && "trying to get non-existent consumer");
    return nullptr;
  }

  virtual ~TraceManager() {
    for (auto &consumerPair : consumers) {
      delete consumerPair.second;
    }
  }
private:
  std::vector<std::pair<hipStream_t, TraceConsumer*>> consumers;
};
char * TraceConsumer::debugdata = nullptr;

TraceManager __trace_manager;

/*******************************************************************************
 * C Interface
 */

extern "C" {
  
  void ___cuprof_accdat_ctor() {
    if (!___cuprof_accdat_var) {
      ___cuprof_accdat_var = (char *) malloc(sizeof(char));
    }
  }
  
  void ___cuprof_accdat_dtor() {
    if (___cuprof_accdat_var) {
      free(___cuprof_accdat_var);
      ___cuprof_accdat_var = NULL;
    }
  }

  void ___cuprof_accdat_append(const char * data, uint64_t data_len) {
    char * var_tmp = (char *) realloc(___cuprof_accdat_var,
                                      ___cuprof_accdat_varlen + data_len + 1);
    if (!var_tmp) {
      fprintf(stderr, "cuprof: Failed to initialize memory access data!\n");
      abort();
    }

    memcpy(var_tmp + ___cuprof_accdat_varlen, data, data_len);
    var_tmp[___cuprof_accdat_varlen + data_len] = '\0';
    
    ___cuprof_accdat_var = var_tmp;
    ___cuprof_accdat_varlen += data_len;
  }

  
  void __trace_fill_info(const void *info, hipStream_t stream) {
    auto *consumer = __trace_manager.getConsumer(stream);
    consumer->fillTraceinfo((traceinfo_t*) info);
  }

  void __trace_copy_to_symbol(hipStream_t stream, const void* symbol, const void *info) {
    //printf("hipMemcpyToSymbol(HIP_SYMBOL(%p), %p, %zu, 0, hipMemcpyHostToDevice)\n", symbol, info, sizeof(traceinfo_t));
    cudaChecked(hipMemcpyToSymbolAsync(HIP_SYMBOL(symbol), info, sizeof(traceinfo_t), 0, hipMemcpyHostToDevice, stream));
  }

  static void __trace_start_callback(hipStream_t stream, hipError_t status, void *vargs);
  static void __trace_stop_callback(hipStream_t stream, hipError_t status, void *vargs);

  //void __trace_touch(hipStream_t stream, const char *header_info) {
  void __trace_touch(hipStream_t stream, uint8_t* sm_filter, uint64_t* cta_filter,
                     uint32_t* warp_filter, size_t* filter_size) {
/*
    printf("%p, %p, %p, %p\n", sm_filter, cta_filter, warp_filter, filter_size);
    
    for (int i = 0; i < filter_size[0]; i++) {
      printf("%u/", sm_filter[i]);
    }
    putchar('\n');
    
    for (int i = 0; i < filter_size[1]; i++) {
      printf("%lu/", cta_filter[i]);
    }
    putchar('\n');
    
    for (int i = 0; i < filter_size[2]; i++) {
      printf("%u/", warp_filter[i]);
    }
    putchar('\n');
*/
    __trace_manager.touchConsumer(stream, (trace_filter_arg_t){sm_filter, cta_filter, warp_filter, filter_size});
  }

  void __trace_start(hipStream_t stream, const char *kernel_name, uint16_t block_size) {
    kernel_trace_arg_t *arg = (kernel_trace_arg_t *) malloc(sizeof(kernel_trace_arg_t));
    if (arg == nullptr) {
      printf("unable to allocate memory\n");
      abort();
    }
    *arg = (kernel_trace_arg_t){kernel_name, block_size};
    cudaChecked(hipStreamAddCallback(stream,
                                      __trace_start_callback, (void*)arg, 0));
  }

  void __trace_stop(hipStream_t stream) {
    cudaChecked(hipStreamAddCallback(stream,
                                      __trace_stop_callback, (void*)nullptr, 0));
  }

/***********************************************************
 * private parts of implementation
 */

  static void __trace_start_callback(hipStream_t stream, hipError_t status, void *vargs) {
    auto *consumer = __trace_manager.getConsumer(stream);
    kernel_trace_arg_t *vargs_cast = (kernel_trace_arg_t *)vargs;
    consumer->start(vargs_cast->kernel_name, vargs_cast->kernel_block_size);
    free(vargs_cast);
  }

  static void __trace_stop_callback(hipStream_t stream, hipError_t status, void *vargs) {
    auto *consumer = __trace_manager.getConsumer(stream);
    consumer->stop();
  }

}
